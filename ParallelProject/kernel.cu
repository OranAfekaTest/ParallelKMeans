#include "hip/hip_runtime.h"
#include "Header.h"


__global__ void moveKernel(Point* points, int N, double dT)
{
	const int i = blockIdx.x * MAX_NUM_OF_THREADS + threadIdx.x;
	if (i < N)
	{
		points[i].x += dT * points[i].Vx;
		points[i].y += dT * points[i].Vy;
		points[i].clusterID = 0;
	}
}

__global__ void groupKernel(Cluster* clusters, int K, Point* points, int N, char* flag)
{
	const int Pi = blockIdx.x * MAX_NUM_OF_THREADS + threadIdx.x;
	int Ci;
	double newDistance, oldDistance;
	//chande to redu[K][3][N] to overcome parallel ++ bug
	if(Pi  < N)
	{
		for (Ci = 0; Ci < K; Ci++)
		{
			
				oldDistance = sqrt(pow(points[Pi].x - clusters[points[Pi].clusterID].centerX, 2) + pow(points[Pi].y - clusters[points[Pi].clusterID].centerY, 2));
				newDistance = sqrt(pow(points[Pi].x - clusters[Ci].centerX, 2) + pow(points[Pi].y - clusters[Ci].centerY, 2));
				//calculate distances of points from cluster centers and make switch if necessery
				if (newDistance < oldDistance)
				{
					*flag = 1;
					points[Pi].clusterID = clusters[Ci].ID;
				}
			}
		
	}
}

__global__ void calcDiametersKernel(Cluster* clusters, int K, Point* points, int N, double* max)
{
	
	const int tid = blockIdx.x * MAX_NUM_OF_THREADS + threadIdx.x;
	int i;
	double distance;

	if (tid < N)
	{
		for (i = tid + 1; i < N; i++) // looking for the distances of the farthest 2 points inside a cluster
			if (points[tid].clusterID == points[i].clusterID)
			{
				distance = sqrt(pow(points[tid].x - points[i].x, 2) + pow(points[tid].y - points[i].y, 2));
				if (clusters[points[tid].clusterID].diameter < distance)
					clusters[points[tid].clusterID].diameter = distance;
					//max[i*K + points[i].clusterID] = distance;
				
					
			}
	}
}

__global__ void getMaxKernel(Cluster* clusters, int K, int N, double* max)
{
	const int tid = threadIdx.x;
	int i;
	double distance;

	for (i = 0; i < N; i++)
	{
		distance = max[i*K + tid];
		if (clusters[tid].diameter < distance)
			clusters[tid].diameter = distance;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t cudaRecalculatePoints(Point* points, int N, double dT)
{
	Point *dev_points = 0;
    hipError_t cudaStatus;
	int blocks = N / MAX_NUM_OF_THREADS + 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaRecalculatePoints - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_points);
		return cudaStatus;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_points, N * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaRecalculatePoints - hipMalloc failed!");
		hipFree(dev_points);
		return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaRecalculatePoints - hipMemcpy failed!");
		hipFree(dev_points);
		return cudaStatus;
    }


    // Launch a kernel on the GPU with one thread for each element.
    moveKernel<<<blocks, MAX_NUM_OF_THREADS >>>(dev_points, N, dT);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "moveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_points);
		return cudaStatus;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaRecalculatePoints - hipDeviceSynchronize returned error code %d after launching moveKernel!\n", cudaStatus);
		hipFree(dev_points);
		return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(points, dev_points, N * sizeof(Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaRecalculatePoints - hipMemcpy failed!");
		hipFree(dev_points);
		return cudaStatus;
    }


    hipFree(dev_points);
    return cudaStatus;
}

hipError_t cudaGroupPoints(Cluster* clusters, int K, Point* points, int N, char* flag)
{
	Cluster* dev_clusters = 0;
	Point *dev_points = 0;
	char* dev_flag = 0;
	hipError_t cudaStatus;
	int blocks = N / MAX_NUM_OF_THREADS + 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, N * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc points failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, K * sizeof(Cluster));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc clusters failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_flag, sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc flag failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input points failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_clusters, clusters, K * sizeof(Cluster), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input clusters failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_flag, flag, sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input flag failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Launch a kernel on the GPU with one thread for each element.
	groupKernel<<<blocks, MAX_NUM_OF_THREADS >>>(dev_clusters, K, dev_points, N, dev_flag);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "groupKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching groupKernel!\n", cudaStatus);
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, N * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy output points failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(flag, dev_flag, sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy output flag failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	hipFree(dev_flag);
	hipFree(dev_points);
	hipFree(dev_clusters);
	return cudaStatus;
}

hipError_t cudaCalcDiameters(Cluster* clusters, int K, Point* points, int N)
{
	Cluster* dev_clusters = 0;
	Point *dev_points = 0;
	double *dev_max = 0;
	//double *max = (double*)calloc(N*K, sizeof(double));
	hipError_t cudaStatus;
	int blocks = N / MAX_NUM_OF_THREADS + 1, i;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, N * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMalloc points failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, K * sizeof(Cluster));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMalloc clusters failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	/*cudaStatus = hipMalloc((void**)&dev_max, K * N * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMalloc clusters failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}*/

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMemcpy input points failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_clusters, clusters, K * sizeof(Cluster), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMemcpy input clusters failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	/*cudaStatus = hipMemcpy(dev_max, max, K * N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMemcpy input max failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}*/

	// Launch a kernel on the GPU with one thread for each element.
	calcDiametersKernel << <blocks, MAX_NUM_OF_THREADS >> >(dev_clusters, K, dev_points, N, dev_max);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "calcDiametersKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calcDiametersKernel!\n", cudaStatus);
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	//getMaxKernel <<<1, K >>>(dev_clusters, K, N, dev_max);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getMaxKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching getMaxKernel!\n", cudaStatus);
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(clusters, dev_clusters, K * sizeof(Cluster), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaCalcDiameters - hipMemcpy output clusters failed!");
		hipFree(dev_max);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}
	
	hipFree(dev_max);
	hipFree(dev_points);
	hipFree(dev_clusters);
	return cudaStatus;
}